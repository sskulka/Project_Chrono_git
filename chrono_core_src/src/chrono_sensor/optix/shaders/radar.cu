#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Han Wang, Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a radar
//
// =============================================================================

#include "chrono_sensor/optix/shaders/device_utils.h"
#include "chrono_sensor/optix/ChOptixDefinitions.h"

extern "C" __global__ void __raygen__radar() {
    const RaygenParameters* raygen = (RaygenParameters*)optixGetSbtDataPointer();
    const RadarParameters& radar = raygen->specific.radar;

    const uint3 idx = optixGetLaunchIndex();
    const uint3 screen = optixGetLaunchDimensions();
    const unsigned int image_index = screen.x * idx.y + idx.x;

    float2 d = (make_float2(idx.x, idx.y) + make_float2(0.5, 0.5)) / make_float2(screen.x, screen.y) * 2.f -
               make_float2(1.f);  //[-1,1]
    float theta = d.x * radar.hFOV / 2.0;
    float phi = -radar.vFOV / 2 + (d.y * .5 + .5) * (radar.vFOV);
    float xy_proj = cos(phi);
    float z = sin(phi);
    float y = xy_proj * sin(theta);
    float x = xy_proj * cos(theta);

    const float t_frac = idx.x / (float)screen.x;
    const float t_traverse = raygen->t0 + t_frac * (raygen->t1 - raygen->t0);  // simulation time when ray is sent
    float3 ray_origin = lerp(raygen->pos0, raygen->pos1, t_frac);
    float4 ray_quat = nlerp(raygen->rot0, raygen->rot1, t_frac);
    float3 forward;
    float3 left;
    float3 up;
    basis_from_quaternion(ray_quat, forward, left, up);
    float3 ray_direction = normalize(forward * x + left * y + up * z);

    PerRayData_radar prd_radar = default_radar_prd();
    unsigned int opt1;
    unsigned int opt2;
    pointer_as_ints(&prd_radar, opt1, opt2);
    unsigned int raytype = (unsigned int)RADAR_RAY_TYPE;
    optixTrace(params.root, ray_origin, ray_direction, radar.clip_near, 1.5f * radar.max_distance, t_traverse,
               OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 0u, 1u, 0u, opt1, opt2,raytype);
    
    float3 vel_global;
    // removing stationary object ray hits
    if (abs(prd_radar.velocity.x) > 0 || abs(prd_radar.velocity.y) > 0 || abs(prd_radar.velocity.z) > 0){
        vel_global = prd_radar.velocity - radar.velocity;

    } else{
        vel_global = make_float3(0,0,0);
    }

    float3 vel_radar_frame =  make_float3(Dot(forward, vel_global), Dot(left, vel_global), Dot(up, vel_global));

    int hIndex = image_index % screen.x;
    int vIndex = image_index / screen.x;

    float azimuth = (hIndex / (float)(screen.x)) * radar.hFOV - radar.hFOV / 2.;
    float elevation = (vIndex / (float)(screen.y)) * (radar.vFOV) - radar.vFOV / 2;
        
    radar.frame_buffer[8 * image_index] = prd_radar.range;
    radar.frame_buffer[8 * image_index + 1] = azimuth;
    radar.frame_buffer[8 * image_index + 2] = elevation; // x velocity
    radar.frame_buffer[8 * image_index + 3] = vel_radar_frame.x; // y velocity
    radar.frame_buffer[8 * image_index + 4] = vel_radar_frame.y; // z velocity
    radar.frame_buffer[8 * image_index + 5] = vel_radar_frame.z; // z velocity
    radar.frame_buffer[8 * image_index + 6] = prd_radar.rcs;
    radar.frame_buffer[8 * image_index + 7] = prd_radar.objectId; // objectId
//    printf("%f %f\n", prd_radar.range, azimuth);

}
